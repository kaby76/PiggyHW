/**
 * Vector sum: C = A + B.
 */

#ifndef __CUDACC__
#define __CUDACC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>


#include "float.h"
#include <builtin_types.h>

// System includes
#include <stdio.h>
#include <assert.h>


extern "C" {

    /**
     * Vector sum on the GPU: C = A + B
     */
    __global__ void VectorSumParallel(float *A, float *B, float *C, int n)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < n)
        {
            C[i] = A[i] + B[i];
        }
    }
}
